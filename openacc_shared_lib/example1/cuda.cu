
#include <hip/hip_runtime.h>
#include <cstdio>
__host__ __device__ void foo() {
#ifdef __CUDA_ARCH__
  std::printf("hello from device\n");
#else
  std::printf("hello from host\n");
#endif
}
