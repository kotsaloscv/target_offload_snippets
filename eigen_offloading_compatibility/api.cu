#include "hip/hip_runtime.h"
#include "api.h"

__host__ __device__
void print_solution(VecType x, int dim)
{
    printf("\n");
    printf("x = ");
    for (int i = 0; i < dim; i++)
        printf("%.3f ", x(i));
    printf("\n\n");
}

__global__
void cuda_kernel(MatType *A, VecType *b, int dim)
{
    VecType x = A->partialPivLu().solve(*b);
    printf("[CUDA]");
    print_solution(x,dim);
}

void solve_cuda(MatType *A, VecType *b, int dim)
{
    cuda_kernel<<<1, 1>>>(A, b, dim);
    hipDeviceSynchronize();
}

__device__ 
void solve_offload(MatType *A, VecType *b, int dim)
{
    VecType x = A->partialPivLu().solve(*b);
    #if defined(USE_OPENACC)
    printf("[OpenACC]");
    #endif
    #if defined(USE_OPENMP)
    printf("[OpenMP]");
    #endif
    print_solution(x,dim);
}
